#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <ATen/cuda/HIPContext.h>
#include <torch/torch.h>

#include <cstdio>
#include <stdint.h>
#include <stdexcept>
#include <limits>

#include "pcg32.h"

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be a contiguous tensor")
#define CHECK_IS_INT(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Int, #x " must be an int tensor")
#define CHECK_IS_FLOATING(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Float || x.scalar_type() == at::ScalarType::Half || x.scalar_type() == at::ScalarType::Double, #x " must be a floating tensor")


inline constexpr __device__ float SQRT3() { return 1.7320508075688772f; }
inline constexpr __device__ float RSQRT3() { return 0.5773502691896258f; }
inline constexpr __device__ float PI() { return 3.141592653589793f; }
inline constexpr __device__ float RPI() { return 0.3183098861837907f; }


template <typename T>
__host__ __device__ T div_round_up(T val, T divisor) {
    return (val + divisor - 1) / divisor;
}

inline __host__ __device__ float signf(const float x) {
    return copysignf(1.0, x);
}

inline __host__ __device__ float clamp(const float x, const float min, const float max) {
    return fminf(max, fmaxf(min, x));
}

inline __host__ __device__ void swapf(float& a, float& b) {
    float c = a; a = b; b = c;
}

inline __device__ int mip_from_pos(const float x, const float y, const float z, const float max_cascade) {
    const float mx = fmaxf(fabsf(x), fmaxf(fabs(y), fabs(z)));
    int exponent;
    frexpf(mx, &exponent); // [0, 0.5) --> -1, [0.5, 1) --> 0, [1, 2) --> 1, [2, 4) --> 2, ...
    return fminf(max_cascade - 1, fmaxf(0, exponent));
}

inline __device__ int mip_from_dt(const float dt, const float H, const float max_cascade) {
    const float mx = dt * H * 0.5;
    int exponent;
    frexpf(mx, &exponent);
    return fminf(max_cascade - 1, fmaxf(0, exponent));
}


////////////////////////////////////////////////////
/////////////           utils          /////////////
////////////////////////////////////////////////////

// rays_o/d: [N, 3]
// nears/fars: [N]
// scalar_t should always be float in use.
template <typename scalar_t>
__global__ void kernel_near_far_from_aabb(
    const scalar_t * __restrict__ rays_o,
    const scalar_t * __restrict__ rays_d,
    const scalar_t * __restrict__ aabb,
    const uint32_t N,
    const float min_near,
    scalar_t * nears, scalar_t * fars
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    rays_o += n * 3;
    rays_d += n * 3;

    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;

    // get near far (assume cube scene)
    float near = (aabb[0] - ox) * rdx;
    float far = (aabb[3] - ox) * rdx;
    if (near > far) swapf(near, far);

    float near_y = (aabb[1] - oy) * rdy;
    float far_y = (aabb[4] - oy) * rdy;
    if (near_y > far_y) swapf(near_y, far_y);

    if (near > far_y || near_y > far) {
        nears[n] = fars[n] = std::numeric_limits<scalar_t>::max();
        return;
    }

    if (near_y > near) near = near_y;
    if (far_y < far) far = far_y;

    float near_z = (aabb[2] - oz) * rdz;
    float far_z = (aabb[5] - oz) * rdz;
    if (near_z > far_z) swapf(near_z, far_z);

    if (near > far_z || near_z > far) {
        nears[n] = fars[n] = std::numeric_limits<scalar_t>::max();
        return;
    }

    if (near_z > near) near = near_z;
    if (far_z < far) far = far_z;

    if (near < min_near) near = min_near;

    nears[n] = near;
    fars[n] = far;
}


void near_far_from_aabb(at::Tensor rays_o, at::Tensor rays_d, at::Tensor aabb, const uint32_t N, const float min_near, at::Tensor nears, at::Tensor fars) {

    static constexpr uint32_t N_THREAD = 256;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "near_far_from_aabb", ([&] {
        kernel_near_far_from_aabb<<<div_round_up(N, N_THREAD), N_THREAD>>>(rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), aabb.data_ptr<scalar_t>(), N, min_near, nears.data_ptr<scalar_t>(), fars.data_ptr<scalar_t>());
    }));
}


// rays_o/d: [N, 3]
// radius: float
// coords: [N, 2]
template <typename scalar_t>
__global__ void kernel_polar_from_ray(
    const scalar_t * __restrict__ rays_o,
    const scalar_t * __restrict__ rays_d,
    const float radius,
    const uint32_t N,
    scalar_t * coords
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    rays_o += n * 3;
    rays_d += n * 3;
    coords += n * 2;

    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;

    // solve t from || o + td || = radius
    const float A = dx * dx + dy * dy + dz * dz;
    const float B = ox * dx + oy * dy + oz * dz; // in fact B / 2
    const float C = ox * ox + oy * oy + oz * oz - radius * radius;

    const float t = (- B + sqrtf(B * B - A * C)) / A; // always use the larger solution (positive)

    // solve theta, phi (assume y is the up axis)
    const float x = ox + t * dx, y = oy + t * dy, z = oz + t * dz;
    const float theta = atan2(sqrtf(x * x + z * z), y); // [0, PI)
    const float phi = atan2(z, x); // [-PI, PI)

    // normalize to [-1, 1]
    coords[0] = 2 * theta * RPI() - 1;
    coords[1] = phi * RPI();
}


void polar_from_ray(at::Tensor rays_o, at::Tensor rays_d, const float radius, const uint32_t N, at::Tensor coords) {

    static constexpr uint32_t N_THREAD = 256;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "polar_from_ray", ([&] {
        kernel_polar_from_ray<<<div_round_up(N, N_THREAD), N_THREAD>>>(rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), radius, N, coords.data_ptr<scalar_t>());
    }));
}


// grid: float, [C, H, H, H]
// N: int, C * H * H * H / 8
// density_thresh: float
// bitfield: uint8, [N]
template <typename scalar_t>
__global__ void kernel_packbits(
    const scalar_t * __restrict__ grid,
    const uint32_t N,
    const float density_thresh,
    uint8_t * bitfield
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    grid += n * 8;

    uint8_t bits = 0;

    #pragma unroll
    for (uint8_t i = 0; i < 8; i++) {
        bits |= grid[i] > density_thresh ? ((uint8_t)1 << i) : 0;
    }

    bitfield[n] = bits;
}


void packbits(at::Tensor grid, const uint32_t N, const float density_thresh, at::Tensor bitfield) {

    static constexpr uint32_t N_THREAD = 256;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grid.scalar_type(), "packbits", ([&] {
        kernel_packbits<<<div_round_up(N, N_THREAD), N_THREAD>>>(grid.data_ptr<scalar_t>(), N, density_thresh, bitfield.data_ptr<uint8_t>());
    }));
}

////////////////////////////////////////////////////
/////////////         training         /////////////
////////////////////////////////////////////////////

// rays_o/d: [N, 3]
// grid: [CHHH / 8]
// xyzs, dirs, deltas: [M, 3], [M, 3], [M, 2]
// dirs: [M, 3]
// rays: [N, 3], idx, offset, num_steps
template <typename scalar_t>
__global__ void kernel_march_rays_train(
    const scalar_t * __restrict__ rays_o,
    const scalar_t * __restrict__ rays_d,  
    const uint8_t * __restrict__ grid,
    const float bound,
    const float dt_gamma, const uint32_t max_steps,
    const uint32_t N, const uint32_t C, const uint32_t H, const uint32_t M,
    const scalar_t* __restrict__ nears, 
    const scalar_t* __restrict__ fars,
    scalar_t * xyzs, scalar_t * dirs, scalar_t * deltas,
    int * rays,
    int * counter,
    const uint32_t perturb
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    rays_o += n * 3;
    rays_d += n * 3;

    // ray marching
    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;

    const float near = nears[n];
    const float far = fars[n];

    const float dt_min = 2 * SQRT3() / max_steps;
    const float dt_max = 2 * SQRT3() * (1 << (C - 1)) / H;
    
    float t0 = near;
    
    if (perturb) {
        pcg32 rng((uint64_t)n);
        t0 += dt_min * rng.next_float();
    }
    
    // first pass: estimation of num_steps
    float t = t0;
    uint32_t num_steps = 0;

    //if (t < far) printf("valid ray %d t=%f near=%f far=%f \n", n, t, near, far);
    
    while (t < far && num_steps < max_steps) {
        // current point
        const float x = clamp(ox + t * dx, -bound, bound);
        const float y = clamp(oy + t * dy, -bound, bound);
        const float z = clamp(oz + t * dz, -bound, bound);

        const float dt = clamp(t * dt_gamma, dt_min, dt_max);

        // get mip level
        const int level = max(mip_from_pos(x, y, z, C), mip_from_dt(dt, H, C)); // range in [0, C - 1]

        const float mip_bound = fminf((float)(1 << level), bound);
        const float mip_rbound = 1 / mip_bound;
        
        // convert to nearest grid position
        const int nx = clamp(0.5 * (x * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int ny = clamp(0.5 * (y * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int nz = clamp(0.5 * (z * mip_rbound + 1) * H, 0.0f, (float)(H - 1));

        const uint32_t index = level * H * H * H + nx * H * H + ny * H + nz;
        const bool occ = grid[index / 8] & (1 << (index % 8));

        // if occpuied, advance a small step, and write to output
        //if (n == 0) printf("t=%f density=%f vs thresh=%f step=%d\n", t, density, density_thresh, num_steps);

        if (occ) {
            num_steps++;
            t += dt;
        // else, skip a large step (basically skip a voxel grid)
        } else {
            // calc distance to next voxel
            const float tx = (((nx + 0.5f + 0.5f * signf(dx)) / (H - 1) * 2 - 1) * mip_bound - x) * rdx;
            const float ty = (((ny + 0.5f + 0.5f * signf(dy)) / (H - 1) * 2 - 1) * mip_bound - y) * rdy;
            const float tz = (((nz + 0.5f + 0.5f * signf(dz)) / (H - 1) * 2 - 1) * mip_bound - z) * rdz;
            const float tt = t + fmaxf(0.0f, fminf(tx, fminf(ty, tz)));
            // step until next voxel
            do { 
                t += clamp(t * dt_gamma, dt_min, dt_max);
            } while (t < tt);
        }
    }

    //printf("[n=%d] num_steps=%d, near=%f, far=%f, dt=%f, max_steps=%f\n", n, num_steps, near, far, dt_min, (far - near) / dt_min);

    // second pass: really locate and write points & dirs
    uint32_t point_index = atomicAdd(counter, num_steps);
    uint32_t ray_index = atomicAdd(counter + 1, 1);
    
    //printf("[n=%d] num_steps=%d, point_index=%d, ray_index=%d\n", n, num_steps, point_index, ray_index);

    // write rays
    rays[ray_index * 3] = n;
    rays[ray_index * 3 + 1] = point_index;
    rays[ray_index * 3 + 2] = num_steps;

    if (num_steps == 0) return;
    if (point_index + num_steps >= M) return;

    xyzs += point_index * 3;
    dirs += point_index * 3;
    deltas += point_index * 2;

    t = t0;
    uint32_t step = 0;

    float last_t = t;

    while (t < far && step < num_steps) {
        // current point
        const float x = clamp(ox + t * dx, -bound, bound);
        const float y = clamp(oy + t * dy, -bound, bound);
        const float z = clamp(oz + t * dz, -bound, bound);

        const float dt = clamp(t * dt_gamma, dt_min, dt_max);

        // get mip level
        const int level = max(mip_from_pos(x, y, z, C), mip_from_dt(dt, H, C)); // range in [0, C - 1]

        const float mip_bound = fminf((float)(1 << level), bound);
        const float mip_rbound = 1 / mip_bound;
        
        // convert to nearest grid position
        const int nx = clamp(0.5 * (x * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int ny = clamp(0.5 * (y * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int nz = clamp(0.5 * (z * mip_rbound + 1) * H, 0.0f, (float)(H - 1));

        // query grid
        const uint32_t index = level * H * H * H + nx * H * H + ny * H + nz;
        const bool occ = grid[index / 8] & (1 << (index % 8));

        // if occpuied, advance a small step, and write to output
        if (occ) {
            // write step
            xyzs[0] = x;
            xyzs[1] = y;
            xyzs[2] = z;
            dirs[0] = dx;
            dirs[1] = dy;
            dirs[2] = dz;
            t += dt;
            deltas[0] = dt;
            deltas[1] = t - last_t; // used to calc depth
            last_t = t;
            xyzs += 3;
            dirs += 3;
            deltas += 2;
            step++;
        // else, skip a large step (basically skip a voxel grid)
        } else {
            // calc distance to next voxel
            const float tx = (((nx + 0.5f + 0.5f * signf(dx)) / (H - 1) * 2 - 1) * mip_bound - x) * rdx;
            const float ty = (((ny + 0.5f + 0.5f * signf(dy)) / (H - 1) * 2 - 1) * mip_bound - y) * rdy;
            const float tz = (((nz + 0.5f + 0.5f * signf(dz)) / (H - 1) * 2 - 1) * mip_bound - z) * rdz;
            const float tt = t + fmaxf(0.0f, fminf(tx, fminf(ty, tz)));
            // step until next voxel
            do { 
                t += clamp(t * dt_gamma, dt_min, dt_max); 
            } while (t < tt);
        }
    }
}


// sigmas: [M]
// rgbs: [M, 3]
// deltas: [M, 2]
// rays: [N, 3], idx, offset, num_steps
// weights_sum: [N], final pixel alpha
// depth: [N,]
// image: [N, 3]
template <typename scalar_t>
__global__ void kernel_composite_rays_train_forward(
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ rgbs,  
    const scalar_t * __restrict__ deltas,
    const int * __restrict__ rays,
    const uint32_t M, const uint32_t N,
    scalar_t * weights_sum,
    scalar_t * depth,
    scalar_t * image
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate 
    uint32_t index = rays[n * 3];
    uint32_t offset = rays[n * 3 + 1];
    uint32_t num_steps = rays[n * 3 + 2];

    // empty ray, or ray that exceed max step count.
    if (num_steps == 0 || offset + num_steps >= M) {
        weights_sum[index] = 0;
        depth[index] = 0;
        image[index * 3] = 0;
        image[index * 3 + 1] = 0;
        image[index * 3 + 2] = 0;
        return;
    }

    sigmas += offset;
    rgbs += offset * 3;
    deltas += offset * 2;

    // accumulate 
    uint32_t step = 0;

    scalar_t T = 1.0f;
    scalar_t r = 0, g = 0, b = 0, ws = 0, t = 0, d = 0;

    while (step < num_steps) {

        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);
        const scalar_t weight = alpha * T;

        // minimal remained transmittence
        //if (weight < 1e-4f) break;

        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];

        t += deltas[1]; // real delta
        d += weight * t;

        ws += weight;

        T *= 1.0f - alpha;

        //printf("[n=%d] num_steps=%d, alpha=%f, w=%f, T=%f, sum_dt=%f, d=%f\n", n, step, alpha, weight, T, sum_delta, d);

        // locate
        sigmas++;
        rgbs += 3;
        deltas += 2;

        step++;
    }

    //printf("[n=%d] rgb=(%f, %f, %f), d=%f\n", n, r, g, b, d);

    // write
    weights_sum[index] = ws; // weights_sum
    depth[index] = d;
    image[index * 3] = r;
    image[index * 3 + 1] = g;
    image[index * 3 + 2] = b;
}


// grad_weights_sum: [N,]
// grad: [N, 3]
// sigmas: [M]
// rgbs: [M, 3]
// deltas: [M, 2]
// rays: [N, 3], idx, offset, num_steps
// weights_sum: [N,], weights_sum here 
// image: [N, 3]
// grad_sigmas: [M]
// grad_rgbs: [M, 3]
template <typename scalar_t>
__global__ void kernel_composite_rays_train_backward(
    const scalar_t * __restrict__ grad_weights_sum,
    const scalar_t * __restrict__ grad_image,
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ rgbs, 
    const scalar_t * __restrict__ deltas,
    const int * __restrict__ rays,
    const scalar_t * __restrict__ weights_sum,
    const scalar_t * __restrict__ image,
    const uint32_t M, const uint32_t N,
    scalar_t * grad_sigmas,
    scalar_t * grad_rgbs
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate 
    uint32_t index = rays[n * 3];
    uint32_t offset = rays[n * 3 + 1];
    uint32_t num_steps = rays[n * 3 + 2];

    if (num_steps == 0 || offset + num_steps >= M) return;

    grad_weights_sum += index;
    grad_image += index * 3;
    weights_sum += index;
    image += index * 3;
    sigmas += offset;
    rgbs += offset * 3;
    deltas += offset * 2;
    grad_sigmas += offset;
    grad_rgbs += offset * 3;

    // accumulate 
    uint32_t step = 0;
    
    scalar_t T = 1.0f;
    const scalar_t r_final = image[0], g_final = image[1], b_final = image[2], ws_final = weights_sum[0];
    scalar_t r = 0, g = 0, b = 0, ws = 0;

    while (step < num_steps) {
        
        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);
        const scalar_t weight = alpha * T;

        //if (weight < 1e-4f) break;

        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];
        ws += weight;

        T *= 1.0f - alpha;

        // write grad_image
        grad_rgbs[0] = grad_image[0] * weight;
        grad_rgbs[1] = grad_image[1] * weight;
        grad_rgbs[2] = grad_image[2] * weight;

        // not grad_sigmas! just a trick to pre-calculate grad_sigmas.
        grad_sigmas[0] = deltas[0] * (
            grad_image[0] * (T * rgbs[0] - (r_final - r)) + 
            grad_image[1] * (T * rgbs[1] - (g_final - g)) + 
            grad_image[2] * (T * rgbs[2] - (b_final - b)) +
            grad_weights_sum[0] * (T - (ws_final - ws))
        );

        //printf("[n=%d] num_steps=%d, T=%f, grad_sigmas=%f, r_final=%f, r=%f\n", n, step, T, grad_sigmas[0], r_final, r);
    
        // locate
        sigmas++;
        rgbs += 3;
        deltas += 2;
        grad_sigmas++;
        grad_rgbs += 3;

        step++;
    }
}


void march_rays_train(at::Tensor rays_o, at::Tensor rays_d, at::Tensor grid, const float bound, const float dt_gamma, const uint32_t max_steps, const uint32_t N, const uint32_t C, const uint32_t H, const uint32_t M, at::Tensor nears, at::Tensor fars, at::Tensor xyzs, at::Tensor dirs, at::Tensor deltas, at::Tensor rays, at::Tensor counter, const uint32_t perturb) {

    static constexpr uint32_t N_THREAD = 256;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "march_rays_train", ([&] {
        kernel_march_rays_train<<<div_round_up(N, N_THREAD), N_THREAD>>>(rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), grid.data_ptr<uint8_t>(), bound, dt_gamma, max_steps, N, C, H, M, nears.data_ptr<scalar_t>(), fars.data_ptr<scalar_t>(), xyzs.data_ptr<scalar_t>(), dirs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), rays.data_ptr<int>(), counter.data_ptr<int>(), perturb);
    }));
}


void composite_rays_train_forward(at::Tensor sigmas, at::Tensor rgbs, at::Tensor deltas, at::Tensor rays, const uint32_t M, const uint32_t N, at::Tensor weights_sum, at::Tensor depth, at::Tensor image) {

    static constexpr uint32_t N_THREAD = 256;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    sigmas.scalar_type(), "composite_rays_train_forward", ([&] {
        kernel_composite_rays_train_forward<<<div_round_up(N, N_THREAD), N_THREAD>>>(sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), rays.data_ptr<int>(), M, N, weights_sum.data_ptr<scalar_t>(), depth.data_ptr<scalar_t>(), image.data_ptr<scalar_t>());
    }));
}


void composite_rays_train_backward(at::Tensor grad_weights_sum, at::Tensor grad_image, at::Tensor sigmas, at::Tensor rgbs, at::Tensor deltas, at::Tensor rays, at::Tensor weights_sum, at::Tensor image, const uint32_t M, const uint32_t N, at::Tensor grad_sigmas, at::Tensor grad_rgbs) {

    static constexpr uint32_t N_THREAD = 256;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grad_image.scalar_type(), "composite_rays_train_backward", ([&] {
        kernel_composite_rays_train_backward<<<div_round_up(N, N_THREAD), N_THREAD>>>(grad_weights_sum.data_ptr<scalar_t>(), grad_image.data_ptr<scalar_t>(), sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), rays.data_ptr<int>(), weights_sum.data_ptr<scalar_t>(), image.data_ptr<scalar_t>(), M, N, grad_sigmas.data_ptr<scalar_t>(), grad_rgbs.data_ptr<scalar_t>());
    }));
}


////////////////////////////////////////////////////
/////////////          infernce        /////////////
////////////////////////////////////////////////////

template <typename scalar_t>
__global__ void kernel_march_rays(
    const uint32_t n_alive, 
    const uint32_t n_step, 
    const int* __restrict__ rays_alive, 
    const scalar_t* __restrict__ rays_t, 
    const scalar_t* __restrict__ rays_o, 
    const scalar_t* __restrict__ rays_d, 
    const float bound,
    const float dt_gamma, const uint32_t max_steps,
    const uint32_t C, const uint32_t H,
    const uint8_t * __restrict__ grid,
    const scalar_t* __restrict__ nears,
    const scalar_t* __restrict__ fars,
    scalar_t* xyzs, scalar_t* dirs, scalar_t* deltas,
    const uint32_t perturb
) {
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= n_alive) return;

    const int index = rays_alive[n]; // ray id
    float t = rays_t[n]; // current ray's t

    // locate
    rays_o += index * 3;
    rays_d += index * 3;
    xyzs += n * n_step * 3;
    dirs += n * n_step * 3;
    deltas += n * n_step * 2;

    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;
    const float near = nears[index], far = fars[index];

    const float dt_min = 2 * SQRT3() / max_steps;
    const float dt_max = 2 * SQRT3() * (1 << (C - 1)) / H;

    // march for n_step steps, record points
    uint32_t step = 0;

    // introduce some randomness (pass in spp as perturb here)
    if (perturb) {
        pcg32 rng((uint64_t)n, (uint64_t)perturb);
        t += dt_min * rng.next_float();
    }

    float last_t = t;

    while (t < far && step < n_step) {
        // current point
        const float x = clamp(ox + t * dx, -bound, bound);
        const float y = clamp(oy + t * dy, -bound, bound);
        const float z = clamp(oz + t * dz, -bound, bound);

        const float dt = clamp(t * dt_gamma, dt_min, dt_max);

        // get mip level
        const int level = max(mip_from_pos(x, y, z, C), mip_from_dt(dt, H, C)); // range in [0, C - 1]

        const float mip_bound = fminf((float)(1 << level), bound);
        const float mip_rbound = 1 / mip_bound;
        
        // convert to nearest grid position
        const int nx = clamp(0.5 * (x * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int ny = clamp(0.5 * (y * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int nz = clamp(0.5 * (z * mip_rbound + 1) * H, 0.0f, (float)(H - 1));

        const uint32_t index = level * H * H * H + nx * H * H + ny * H + nz;
        const bool occ = grid[index / 8] & (1 << (index % 8));

        // if occpuied, advance a small step, and write to output
        if (occ) {
            // write step
            xyzs[0] = x;
            xyzs[1] = y;
            xyzs[2] = z;
            dirs[0] = dx;
            dirs[1] = dy;
            dirs[2] = dz;
            // calc dt
            t += dt;
            deltas[0] = dt;
            deltas[1] = t - last_t; // used to calc depth
            last_t = t;
            // step
            xyzs += 3;
            dirs += 3;
            deltas += 2;
            step++;

        // else, skip a large step (basically skip a voxel grid)
        } else {
            // calc distance to next voxel
            const float tx = (((nx + 0.5f + 0.5f * signf(dx)) / (H - 1) * 2 - 1) * mip_bound - x) * rdx;
            const float ty = (((ny + 0.5f + 0.5f * signf(dy)) / (H - 1) * 2 - 1) * mip_bound - y) * rdy;
            const float tz = (((nz + 0.5f + 0.5f * signf(dz)) / (H - 1) * 2 - 1) * mip_bound - z) * rdz;
            const float tt = t + fmaxf(0.0f, fminf(tx, fminf(ty, tz)));
            // step until next voxel
            do { 
                t += clamp(t * dt_gamma, dt_min, dt_max);
            } while (t < tt);
        }
    }
}


void march_rays(const uint32_t n_alive, const uint32_t n_step, at::Tensor rays_alive, at::Tensor rays_t, at::Tensor rays_o, at::Tensor rays_d, const float bound, const float dt_gamma, const uint32_t max_steps, const uint32_t C, const uint32_t H, at::Tensor grid, at::Tensor near, at::Tensor far, at::Tensor xyzs, at::Tensor dirs, at::Tensor deltas, const uint32_t perturb) {
    static constexpr uint32_t N_THREAD = 256;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "march_rays", ([&] {
        kernel_march_rays<<<div_round_up(n_alive, N_THREAD), N_THREAD>>>(n_alive, n_step, rays_alive.data_ptr<int>(), rays_t.data_ptr<scalar_t>(), rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), bound, dt_gamma, max_steps, C, H, grid.data_ptr<uint8_t>(), near.data_ptr<scalar_t>(), far.data_ptr<scalar_t>(), xyzs.data_ptr<scalar_t>(), dirs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), perturb);
    }));
}


template <typename scalar_t>
__global__ void kernel_composite_rays(
    const uint32_t n_alive, 
    const uint32_t n_step, 
    const int* __restrict__ rays_alive, 
    scalar_t* rays_t, 
    const scalar_t* __restrict__ sigmas, 
    const scalar_t* __restrict__ rgbs, 
    const scalar_t* __restrict__ deltas, 
    scalar_t* weights_sum, scalar_t* depth, scalar_t* image
) {
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= n_alive) return;

    const int index = rays_alive[n]; // ray id
    scalar_t t = rays_t[n]; // current ray's t

    // locate 
    sigmas += n * n_step;
    rgbs += n * n_step * 3;
    deltas += n * n_step * 2;

    weights_sum += index;
    depth += index;
    image += index * 3;
    
    scalar_t weight_sum = weights_sum[0];
    scalar_t d = depth[0];
    scalar_t r = image[0];
    scalar_t g = image[1];
    scalar_t b = image[2];

    // accumulate 
    uint32_t step = 0;
    while (step < n_step) {
        
        // ray is terminated if delta == 0
        if (deltas[0] == 0) break;
        
        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);

        /* 
        T_0 = 1; T_i = \prod_{j=0}^{i-1} (1 - alpha_j)
        w_i = alpha_i * T_i
        --> 
        T_i = 1 - \sum_{j=0}^{i-1} w_j
        */
        const scalar_t T = 1 - weight_sum;
        const scalar_t weight = alpha * T;
        weight_sum += weight;

        t += deltas[1]; // real delta
        d += weight * t;
        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];

        //printf("[n=%d] num_steps=%d, alpha=%f, w=%f, T=%f, sum_dt=%f, d=%f\n", n, step, alpha, weight, T, sum_delta, d);

        // ray is terminated if T is too small
        if (T < 1e-4) break;

        // locate
        sigmas++;
        rgbs += 3;
        deltas += 2;
        step++;
    }

    //printf("[n=%d] rgb=(%f, %f, %f), d=%f\n", n, r, g, b, d);

    // rays_t = -1 means ray is terminated early.
    if (step < n_step) {
        rays_t[n] = -1;
    } else {
        rays_t[n] = t;
    }

    weights_sum[0] = weight_sum; // this is the thing I needed!
    depth[0] = d;
    image[0] = r;
    image[1] = g;
    image[2] = b;
}


void composite_rays(const uint32_t n_alive, const uint32_t n_step, at::Tensor rays_alive, at::Tensor rays_t, at::Tensor sigmas, at::Tensor rgbs, at::Tensor deltas, at::Tensor weights, at::Tensor depth, at::Tensor image) {
    static constexpr uint32_t N_THREAD = 256;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    image.scalar_type(), "composite_rays", ([&] {
        kernel_composite_rays<<<div_round_up(n_alive, N_THREAD), N_THREAD>>>(n_alive, n_step, rays_alive.data_ptr<int>(), rays_t.data_ptr<scalar_t>(), sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), weights.data_ptr<scalar_t>(), depth.data_ptr<scalar_t>(), image.data_ptr<scalar_t>());
    }));
}


template <typename scalar_t>
__global__ void kernel_compact_rays(
    const uint32_t n_alive, 
    int* rays_alive, 
    const int* __restrict__ rays_alive_old, 
    scalar_t* rays_t, 
    const scalar_t* __restrict__ rays_t_old, 
    int* alive_counter
) {
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= n_alive) return;

    // rays_t_old[n] < 0 means ray died in last composite kernel.
    if (rays_t_old[n] >= 0) {
        const int index = atomicAdd(alive_counter, 1);
        rays_alive[index] = rays_alive_old[n];
        rays_t[index] = rays_t_old[n];
    }
}


void compact_rays(const uint32_t n_alive, at::Tensor rays_alive, at::Tensor rays_alive_old, at::Tensor rays_t, at::Tensor rays_t_old, at::Tensor alive_counter) {
    static constexpr uint32_t N_THREAD = 256;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_t.scalar_type(), "compact_rays", ([&] {
        kernel_compact_rays<<<div_round_up(n_alive, N_THREAD), N_THREAD>>>(n_alive, rays_alive.data_ptr<int>(), rays_alive_old.data_ptr<int>(), rays_t.data_ptr<scalar_t>(), rays_t_old.data_ptr<scalar_t>(), alive_counter.data_ptr<int>());
    }));
}